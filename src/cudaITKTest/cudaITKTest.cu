#include <thrust/device_vector.h>

#include "itkImage.h"
//#include "itkImageFileReader.h"
//#include "itkGDCMImageIO.h"
//#include "itkImageToVTKImageFilter.h"
//#include "itkGDCMImageIOFactory.h"
//#include "itkNrrdImageIOFactory.h"
//#include "itkNiftiImageIOFactory.h"
//#include "itkMINCImageIOFactory.h"
//#include "itkMetaImageIOFactory.h"
//#include "itkBMPImageIOFactory.h"
//#include "itkPNGImageIOFactory.h"
//#include "itkJPEGImageIOFactory.h"
//#include "itkGDCMSeriesFileNames.h"
//#include "itkImageSeriesReader.h"
//#include "itkOrientImageFilter.h"
//#include "itkMetaDataObject.h"
//#include "itkImageFileReader.h"
//
class VolumeImage
{
public:
    VolumeImage();
    ~VolumeImage();
    //itk::Image<float, 3U>::Pointer getITKImagePointer() { return itkImage; }
    void readITKImage();
private:
    itk::Image<float, 3U>::Pointer itkImage;
};

VolumeImage::VolumeImage()
{
}

VolumeImage::~VolumeImage()
{
}

void VolumeImage::readITKImage()
{

}


int main(int argc, char * argv[])
{
    //VolumeImage itkVolumeImage;

    /*
    using PixelType = float;
    using ImageType = itk::Image<PixelType, 3>;
    //1. read the image from local file
    itk::ImageFileReader<ImageType>::Pointer reader = itk::ImageFileReader<ImageType>::New();
    reader->SetFileName(path);
    try {
    reader->Update();
    }
    catch (itk::ExceptionObject & err) {
    std::cout << "ERROR while parsing fixed volume: " << err.what() << std::endl;
    return false;
    }

    //2. OrientImageFilter change image orientation(direction)
    itk::OrientImageFilter<ImageType, ImageType>::Pointer orientationFilter = itk::OrientImageFilter<ImageType, ImageType>::New();
    orientationFilter->UseImageDirectionOn();
    orientationFilter->SetDesiredCoordinateOrientation(itk::SpatialOrientation::ITK_COORDINATE_ORIENTATION_RAI);
    orientationFilter->SetInput(reader->GetOutput());
    try {
    orientationFilter->Update();
    }
    catch (itk::ExceptionObject & err) {
    std::cout << "ERROR while correcting fixed volume orientation: " << err.what() << std::endl;
    return false;
    }

    //3. get image that has been changed orientation
    itk::Image<float, 3U> *volumeImgPtr = orientationFilter->GetOutput();
    */

    //std::string filePath = "D:\\projects\\Fussen\\shangtaoSkull\\dicom\\0262.dcm";

    //using PixelType = signed short;
    //using ImageType = itk::Image<PixelType, 3>;
    ////1. read the image from local file
    //itk::ImageFileReader<ImageType>::Pointer reader = itk::ImageFileReader<ImageType>::New();
    //reader->SetFileName(filePath);
    //try {
    //    reader->Update();
    //}
    //catch (itk::ExceptionObject & err) {
    //    std::cout << "ERROR " << err.what() << std::endl;
    //    return false;
    //}

    //itk::OrientImageFilter<ImageType, ImageType>::Pointer orientationFilter = itk::OrientImageFilter<ImageType, ImageType>::New();
    //orientationFilter->UseImageDirectionOn();
    //orientationFilter->SetDesiredCoordinateOrientation(itk::SpatialOrientation::ITK_COORDINATE_ORIENTATION_RAI);
    //orientationFilter->SetInput(reader->GetOutput());
    //try {
    //    orientationFilter->Update();
    //}
    //catch (itk::ExceptionObject & err) {
    //    std::cout << "ERROR while correcting fixed volume orientation: " << err.what() << std::endl;
    //    return false;
    //}

    ////3. get image that has been changed orientation
    ////itk::Image<PixelType, 3U> *volumeImgPtr = orientationFilter->GetOutput();

    ////������ȡDCM��GDCMIOImage��
    //using ImageIOType = itk::GDCMImageIO;
    //ImageIOType::Pointer gdcmImageIO = ImageIOType::New();
    //reader->SetImageIO(gdcmImageIO);
    //try
    //{
    //    reader->Update();
    //}
    //catch (itk::ExceptionObject& e)
    //{
    //    std::cerr << "exception in file reader" << std::endl;
    //    std::cerr << e << std::endl;
    //    return EXIT_FAILURE;
    //}

    ////�����˲�����ת��ΪVTK����
    //typedef itk::ImageToVTKImageFilter<ImageType> ConnectorType;
    //ConnectorType::Pointer connector = ConnectorType::New();
    //connector->SetInput(reader->GetOutput());
    //try
    //{
    //    connector->Update();
    //}
    //catch (itk::ExceptionObject& e)
    //{
    //    std::cerr << "exception in file reader" << std::endl;
    //    std::cerr << e << std::endl;
    //    return EXIT_FAILURE;
    //}

    ///* vtkImageActor��3D��������Ⱦͼ�� */
    //vtkSmartPointer<vtkImageActor> actor = vtkImageActor::New();
    //actor->SetInputData(connector->GetOutput());
    //actor->InterpolateOff();
    //actor->Update();


    //vtkSmartPointer<vtkRenderer> render = vtkRenderer::New();
    //render->AddActor(actor);
    //render->SetBackground(255, 1, 1);//���ô��ڱ�����ɫ

    //vtkSmartPointer<vtkRenderWindow> window = vtkRenderWindow::New();
    //window->SetSize(600, 600);       //���ô��ڴ�С
    //window->AddRenderer(render);
    //window->SetWindowName("the CT picture ");//���ô�������

    //vtkSmartPointer<vtkRenderWindowInteractor> interactor = vtkRenderWindowInteractor::New();
    //interactor->SetRenderWindow(window);

    //interactor->Initialize();
    //interactor->Start();
    //return 0;

    return EXIT_SUCCESS;


}



